#include "hip/hip_runtime.h"
#include "noise_module_base.cuh"

// Constants used by the current version of libnoise.
const int X_NOISE_GEN = 1619;
const int Y_NOISE_GEN = 31337;
const int Z_NOISE_GEN = 6971;
const int SEED_NOISE_GEN = 1013;
const int SHIFT_NOISE_GEN = 8;

NoiseModule::NoiseModule() {

}

double NoiseModule::GetValue(double x, double y, double z) {
    return 0.0;
}

double NoiseModule::MakeInt32Range(double n) {
    if (n >= 1073741824.0) {
        return (2.0 * fmod (n, 1073741824.0)) - 1073741824.0;
    }
    else if (n <= -1073741824.0) {
        return (2.0 * fmod (n, 1073741824.0)) + 1073741824.0;
    }
    else {
        return n;
    }
}

double NoiseModule::GradientNoise3D(double fx, double fy, double fz, int ix,
                                    int iy, int iz, int seed) {
    // Randomly generate a gradient vector given the integer coordinates of the
    // input value.  This implementation generates a random number and uses it
    // as an index into a normalized-vector lookup table.
    int vectorIndex = ( X_NOISE_GEN    * ix
                        + Y_NOISE_GEN    * iy
                        + Z_NOISE_GEN    * iz
                        + SEED_NOISE_GEN * seed) & 0xffffffff;
    vectorIndex ^= (vectorIndex >> SHIFT_NOISE_GEN);
    vectorIndex &= 0xff;

    double xvGradient = g_randomVectors((vectorIndex << 2)    );
    double yvGradient = g_randomVectors((vectorIndex << 2) + 1);
    double zvGradient = g_randomVectors((vectorIndex << 2) + 2);

    // Set up us another vector equal to the distance between the two vectors
    // passed to this function.
    double xvPoint = (fx - (double)ix);
    double yvPoint = (fy - (double)iy);
    double zvPoint = (fz - (double)iz);

    // Now compute the dot product of the gradient vector with the distance
    // vector.  The resulting value is gradient noise.  Apply a scaling value
    // so that this noise value ranges from -1.0 to 1.0.
    return ((xvGradient * xvPoint)
            + (yvGradient * yvPoint)
            + (zvGradient * zvPoint)) * 2.12;
}

double NoiseModule::GradientCoherentNoise3D(double x, double y, double z,
                                            int seed) {
    // Create a unit-length cube aligned along an integer boundary.  This cube
    // surrounds the input point.
    int x0 = (x > 0.0? (int)x: (int)x - 1);
    int x1 = x0 + 1;
    int y0 = (y > 0.0? (int)y: (int)y - 1);
    int y1 = y0 + 1;
    int z0 = (z > 0.0? (int)z: (int)z - 1);
    int z1 = z0 + 1;

    // Map the difference between the coordinates of the input value and the
    // coordinates of the cube's outer-lower-left vertex onto an S-curve.
    double xs = 0, ys = 0, zs = 0;
    xs = SCurve3 (x - (double)x0);
    ys = SCurve3 (y - (double)y0);
    zs = SCurve3 (z - (double)z0);

    // Now calculate the noise values at each vertex of the cube.  To generate
    // the coherent-noise value at the input point, interpolate these eight
    // noise values using the S-curve value as the interpolant (trilinear
    // interpolation.)
    double n0, n1, ix0, ix1, iy0, iy1;
    n0   = GradientNoise3D (x, y, z, x0, y0, z0, seed);
    n1   = GradientNoise3D (x, y, z, x1, y0, z0, seed);
    ix0  = LinearInterp (n0, n1, xs);
    n0   = GradientNoise3D (x, y, z, x0, y1, z0, seed);
    n1   = GradientNoise3D (x, y, z, x1, y1, z0, seed);
    ix1  = LinearInterp (n0, n1, xs);
    iy0  = LinearInterp (ix0, ix1, ys);
    n0   = GradientNoise3D (x, y, z, x0, y0, z1, seed);
    n1   = GradientNoise3D (x, y, z, x1, y0, z1, seed);
    ix0  = LinearInterp (n0, n1, xs);
    n0   = GradientNoise3D (x, y, z, x0, y1, z1, seed);
    n1   = GradientNoise3D (x, y, z, x1, y1, z1, seed);
    ix1  = LinearInterp (n0, n1, xs);
    iy1  = LinearInterp (ix0, ix1, ys);

    return LinearInterp (iy0, iy1, zs);
}

double NoiseModule::LinearInterp(double n0, double n1, double a) {
    return ((1.0 - a) * n0) + (a * n1);
}

double NoiseModule::SCurve3(double a) {
    return (a * a * (3.0 - 2.0 * a));
}
